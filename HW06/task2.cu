#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "stencil.cuh"

int main(int argc, char* argv[]) {
    unsigned int n = std::atoi(argv[1]);
    unsigned int R = std::atoi(argv[2]);
    unsigned int threads_per_block = std::atoi(argv[3]);

    float* h_image = new float[n];
    float* h_mask = new float[2 * R + 1];
    float* h_output = new float[n];

    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    for (unsigned int i = 0; i < n; i++)
    {
        h_image[i] = dist(generator);
        // h_image[i] = i;
    }

    for (unsigned int i = 0; i < 2 * R + 1; i++)
    {
        h_mask[i] = dist(generator);
        // h_mask[i] = 1;
    }

    float *d_image, *d_mask, *d_output;
    hipMalloc((void**)&d_image, n * sizeof(float));
    hipMalloc((void**)&d_mask, (2 * R + 1) * sizeof(float));
    hipMalloc((void**)&d_output, n * sizeof(float));

    hipMemcpy(d_image, h_image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    stencil(d_image, d_mask, d_output, n, R, threads_per_block);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);


    std::cout << h_output[n - 1] << std::endl;
    std::cout << elapsed_time << std::endl;

 
    delete[] h_image;
    delete[] h_mask;
    delete[] h_output;
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);

    return 0;
}
